
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#define N 500000

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__
void saxpy(int n, float a, float * x, float * y)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < n) y[idx] = a*x[idx] + y[idx];
}

__global__
void kernel_a(float* x, float* y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] += 1;

}

__global__
void kernel_c(float* x, float* y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] -= 1;

}

void initialize(float* h_temp){
  for (int i = 0; i < N; ++i){
    h_temp[i] = (float)i;
  }
}

int main(){

hipStream_t stream1;

hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);

hipblasHandle_t cublas_handle;
hipblasCreate(&cublas_handle);

float* h_x;
float* h_y;

h_x = (float*) malloc(N * sizeof(float));
h_y = (float*) malloc(N * sizeof(float));

printf("yes?\n");
initialize(h_x);
initialize(h_y);
printf("yes?\n");

float* d_x;
float* d_y;
float d_a = 5.0;

hipMalloc((void**) &d_x, N * sizeof(float));
hipMalloc((void**) &d_y, N * sizeof(float));

printf("yes?\n");
hipblasSetVector(N, sizeof(h_x[0]), h_x, 1, d_x, 1); // similar to cudaMemcpy
hipblasSetVector(N, sizeof(h_y[0]), h_y, 1, d_y, 1); // similar to cudaMemcpy
cudaCheckErrors("Mallocing failed");

hipGraph_t graph; // main graph
hipGraph_t libraryGraph; // sub graph for cuBLAS call
printf("yes?\n");
std::vector<hipGraphNode_t> nodeDependencies;
//cudaGraphNode_t nodeDependencies[];
hipGraphNode_t kernelNode1, kernelNode2, libraryNode;

hipKernelNodeParams kernelNode1Params {0};
hipKernelNodeParams kernelNode2Params {0};

void *kernelArgs[2] = {(void *)&d_x, (void *)&d_y};

int threads = 512;
int blocks = (N + (threads - 1) / threads);

kernelNode1Params.func = (void *)kernel_a;
//kernelNode1Params.gridDim = dim3(blocks, 1, 1);
//kernelNode1Params.blockDim = dim3(threads, 1, 1);
kernelNode1Params.gridDim = dim3(1024, 1, 1);
kernelNode1Params.blockDim = dim3(512, 1, 1);
kernelNode1Params.sharedMemBytes = 0;
kernelNode1Params.kernelParams = (void **)kernelArgs;
kernelNode1Params.extra = NULL;
printf("yes!?\n");

//cudaGraphAddKernelNode(&kernelNode1, graph, nodeDependencies.data(),
hipGraphAddKernelNode(&kernelNode1, graph, NULL,
                         0, &kernelNode1Params);
printf("yes!\n");
cudaCheckErrors("Adding kernelNode1 failed");

nodeDependencies.push_back(kernelNode1);

hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);

hipblasSaxpy(cublas_handle, N, &d_a, d_x, 1, d_y, 1);

hipStreamEndCapture(stream1, &libraryGraph);

hipGraphAddChildGraphNode(&libraryNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), libraryGraph);
cudaCheckErrors("Adding libraryNode failed");

nodeDependencies.clear();
nodeDependencies.push_back(libraryNode);

kernelNode2Params.func = (void *)kernel_c;
kernelNode2Params.gridDim = dim3(blocks, 1, 1);
kernelNode2Params.blockDim = dim3(threads, 1, 1);
kernelNode2Params.sharedMemBytes = 0;
kernelNode2Params.kernelParams = (void **)kernelArgs;
kernelNode2Params.extra = NULL;

hipGraphAddKernelNode(&kernelNode2, graph, nodeDependencies.data(),
                         nodeDependencies.size(), &kernelNode2Params);
cudaCheckErrors("Adding kernelNode1 failed");

nodeDependencies.clear();
nodeDependencies.push_back(kernelNode2);

hipGraphNode_t *nodes = NULL;
size_t numNodes = 0;
hipGraphGetNodes(graph, nodes, &numNodes);
cudaCheckErrors("Graph instantiation failed");
printf("Number of the nodes in the graph = %zu\n", numNodes);

hipGraphExec_t instance;
hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
cudaCheckErrors("Graph instantiation failed");

for (int i = 0; i < 100; ++i){
// Launch the graph instance
//printf("launching graph\n");
hipGraphLaunch(instance, stream1);
hipStreamSynchronize(stream1);

}

hipMemcpy(h_y, d_y, N, hipMemcpyDeviceToHost);

hipDeviceSynchronize();

for (int i = 0; i < N; ++i){
//    printf("%2.0f ", h_y[i]);
}
printf("\n");






}
