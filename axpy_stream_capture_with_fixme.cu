#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime_api.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define M 1024
#define N 500000

__global__
void saxpy(int n, float a, float * x, float * y)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < n) y[idx] = a*x[idx] + y[idx];
}

__global__
void kernel_a(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

__global__
void kernel_b(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

__global__
void kernel_c(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

__global__
void kernel_d(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

int main(){

hipEvent_t event1;
hipEvent_t event2;

hipEventCreateWithFlags(&event1, hipEventDisableTiming);
hipEventCreateWithFlags(&event2, hipEventDisableTiming);

const int num_streams = 2;

hipStream_t streams[num_streams];

for (int i = 0; i < num_streams; ++i){
hipStreamCreate(&streams[i]);
}

hipError_t cuda_error;

float* h_x;
float* h_y;

h_x = (float*) malloc(N * sizeof(float));

for (int i = 0; i < N; ++i){
    h_x[i] = (float)i;
//    printf("%2.0f ", h_x[i]);
}
printf("\n");

h_y = (float*) malloc(N * sizeof(float));

for (int i = 0; i < N; ++i){
    h_y[i] = (float)i;
//    printf("%2.0f ", h_y[i]);
}
printf("\n");

float* d_x;
float* d_y;

hipMalloc((void**) &d_x, N * sizeof(float));
hipMalloc((void**) &d_y, N * sizeof(float));

hipMemcpy(d_x, h_x, N, hipMemcpyHostToDevice);
hipMemcpy(d_y, h_y, N, hipMemcpyHostToDevice);

bool graphCreated=false;
hipGraph_t graph;
hipGraphExec_t instance;

checkCudaErrors(hipGraphCreate(&graph, 0));

for (int i = 0; i < 100; ++i){
if (graphCreated == false){
// Starting stream capture
hipStreamBeginCapture(streams[0], hipStreamCaptureModeGlobal);

kernel_a<<<1024, 512, 0, streams[0]>>>(d_x, d_y);

hipEventRecord(event1, streams[0]);

kernel_b<<<1024, 512, 0, streams[0]>>>(d_x, d_y);

hipStreamWaitEvent(streams[1], event1);

kernel_c<<<1024, 512, 0, streams[1]>>>(d_x, d_y);

hipEventRecord(event2, streams[1]);

hipStreamWaitEvent(streams[0], event2);

kernel_d<<<1024, 512, 0, streams[0]>>>(d_x, d_y);

hipStreamEndCapture(streams[0], &graph);

// Creating the graph instance

hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

graphCreated = true;
}
// Launch the graph instance
//printf("launching graph\n");
hipGraphLaunch(instance, streams[0]);
hipStreamSynchronize(streams[0]);

}

hipMemcpy(h_y, d_y, N, hipMemcpyDeviceToHost);

hipDeviceSynchronize();

for (int i = 0; i < N; ++i){
//    printf("%2.0f ", h_y[i]);
}
printf("\n");






}
